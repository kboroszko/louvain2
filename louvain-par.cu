#include "hip/hip_runtime.h"
//
// Created by kajetan on 30.04.2020.
//
extern "C" {
    #include <assert.h>
    #include "louvain.h"
    #include "graph-utils.h"
}

#include "errors.h"
#include <thrust/fill.h>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

//__device__ float atomicAdd(float* address, float val)
//{
//    unsigned int* address_as_ull =
//            (unsigned int*)address;
//    unsigned int old = *address_as_ull, assumed;
//
//    do {
//        assumed = old;
//        old = atomicCAS(address_as_ull, assumed,
//                        __float_as_int(val +
//                                               __int_as_float(assumed)));
//
//        // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
//    } while (assumed != old);
//
//    return __int_as_float(old);
//}

__device__ int moveValid(int from, int to, int* cliqueSizes);

float getKi(Graph *g, int vertice){
    float sum=0;
    for(int i=EDGES_IDX(g,vertice-1); i<EDGES_IDX(g,vertice); i++){
        sum+= g->edges[i].value;
    }
    return sum;
}

float getKiin(Graph *g, int vertice, int* cliques, int in ){
    float sum=0;
    for(int i=EDGES_IDX(g,vertice-1); i<EDGES_IDX(g,vertice); i++){
        Edge e = g->edges[i];
        if(e.to != vertice && cliques[e.to] == in){
            sum+= e.value;
        }
    }
    return sum;
}

int bestClique(Graph *g, int vertice, int *cliques, float*sigmaTots, float m){
    float best = 0;
    int bestClique = -1;
    for(int i=EDGES_IDX(g,vertice-1); i<EDGES_IDX(g,vertice); i++){
        int to = g->edges[i].to;
        int in = cliques[to];
        if(in != bestClique && in != cliques[vertice]){
            float deltaQ =  0;//dQ(g, vertice, cliques, in, sigmaTots, m);
            if(deltaQ > best){
                best = deltaQ;
                bestClique = in;
            } else if (deltaQ == best){
                if(bestClique > in){
                    bestClique = in;
                }
            }
        }
    }
    return bestClique;
}

float selfLoop(Graph *g, int vertice){
    for(int k=EDGES_IDX(g,vertice-1); k<EDGES_IDX(g,vertice); k++){
        Edge e = g->edges[k];
        if(e.to == vertice){
            return e.value;
        }
    }
    return 0.0f;
}

int verticeHasEdges(Graph *g, int vertice){
    int has = EDGES_IDX(g, vertice-1) != EDGES_IDX(g, vertice);
    return has;
}

float modularity(Graph *g, int * cliques){
    float sum = 0;
    float m = 0;
    float * ac = (float*) calloc(g->size, sizeof(float));

    for(int i=0; i<g->size; i++){
        float ki = getKi(g, i);
        int clique = cliques[i];
        ac[clique] += ki;
        m += ki;
    }
    m = m/2.f;

    for(int i=0; i<g->size; i++){
        sum += ac[i] * ac[i];
    }

    sum = -sum/(2.f * m);

    for(int i=0; i < g->size; i++){
        float EiwCiBezi = getKiin(g, i, cliques, cliques[i]);
        sum += EiwCiBezi + selfLoop(g, i);
    }
    free(ac);
    return sum/(2.f*m);
}

int compareMoves( const void * a, const void * b){
    Move* ma = ((Move*)a);
    Move* mb = ((Move*)b);
    if(ma->gain > mb->gain) {
        return -1;
    } else if(mb->gain > ma->gain){
        return 1;
    } else {
        return 0;
    }
}

int compareMovesThrust( Move a, Move b){
    if(a.gain > b.gain) {
        return -1;
    } else if(b.gain > a.gain){
        return 1;
    } else {
        return 0;
    }
}

void applyBestMoves(int* cliques, Move* moves ,int nMoves, int nBest, int sort){
    if(nMoves == 0){
        return;
    }
    assert(nMoves >= nBest);
    if(nMoves != nBest && sort != 0){
        qsort(moves, nMoves, sizeof(Move), compareMoves);
    }
    for(int i=0; i < nBest; i++){
        Move m = moves[i];
        cliques[m.vertice] = m.toClique;
    }
}

float previewModularity(Graph * g, int*newCliques, Move* moves, int nMoves, int nBest, int sort){
    applyBestMoves(newCliques, moves, nMoves, nBest, sort);
    float newMod = modularity(g, newCliques);
    return newMod;
}


__device__ int moveValid(int from, int to, int* cliqueSizes){
    if(from == to){
        return 0;
    }
    if(from > to || cliqueSizes[from] > 1 || cliqueSizes[to] > 1){
        return 1;
    }
    return 0;
}



int calculateMovesToApply(int iters, int movesDone, int nMoves){
    int ret = movesDone;
    for(int i=0; i< iters; i++){
        ret = (ret + 1)/ 2;
    }
    return ret > 0 ? ret : 1;
}

__device__ float getKiDevice(int numEdges, Edge* edges){
    float sum = 0;
    for(int i=0; i<numEdges; i++){
        sum += edges[i].value;
    }
    return sum;
}

void copyGraphToDevice(Graph*g, Graph**deviceGraphPtr){

    Edge * edgesPtr ;
    int * vertPtr ;

    HANDLE_ERROR(hipMalloc((void**) &edgesPtr, sizeof(Edge) * g->numEdges));
    HANDLE_ERROR(hipMalloc((void**) &vertPtr, sizeof(int) * g->size));

//    printf("graph tables malloc succeded\n");


    HANDLE_ERROR(hipMemcpy((void*) edgesPtr, (void*)g->edges, sizeof(Edge) * g->numEdges, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy((void*) vertPtr, (void*)g->verticeLastEdgeExclusive, sizeof(int) * g->size, hipMemcpyHostToDevice));

//    printf("copying succeded\n");


    HANDLE_ERROR(hipMalloc((void**)deviceGraphPtr, sizeof(Graph)));

    Graph gr = {.size=g->size, .numEdges=g->numEdges, .edges=edgesPtr, .verticeLastEdgeExclusive=vertPtr};

    HANDLE_ERROR(hipMemcpy((void*)*deviceGraphPtr, (void*)&gr, sizeof(Graph), hipMemcpyHostToDevice));

//    printf("graph init succeded\n");
}

__device__ float getKiinDevice(Graph *g, int vertice, int* cliques, int in ){
    float sum=0;
    for(int i=EDGES_IDX(g,vertice-1); i<EDGES_IDX(g,vertice); i++){
        Edge e = g->edges[i];
        if(e.to != vertice && cliques[e.to] == in){
            sum+= e.value;
        }
    }
    return sum;
}



__device__ float dQDevice(Graph*g, int vertice, int *cliques, int in, float* sigmaTot, float m, int numEdges, Edge* edges){

    float ki = getKiDevice(numEdges, edges);
    float kiin = getKiinDevice(g, vertice, cliques, in);
    float EiwCiBezi = getKiinDevice(g, vertice, cliques, cliques[vertice]);
    float aciBezi= sigmaTot[cliques[vertice]] - ki;
    float acj = sigmaTot[in];
    float part1 = (kiin - EiwCiBezi)/m;
    float part2 = ki * (aciBezi - acj)/(2 * m * m);
    return  part1+part2;
}




void copyArrayToDevice(int * arr, int size, int** deviceArray){
    HANDLE_ERROR(hipMalloc((void**) deviceArray, sizeof(int) * size));
    HANDLE_ERROR(hipMemcpy((void*) *deviceArray, (void*)arr, sizeof(int) * size, hipMemcpyHostToDevice));
}

void copyFloatArrayToDevice(float * arr, int size, float** deviceArray){
    HANDLE_ERROR(hipMalloc((void**) deviceArray, sizeof(float) * size));
    HANDLE_ERROR(hipMemcpy((void*) *deviceArray, (void*)arr, sizeof(float) * size, hipMemcpyHostToDevice));
}

__global__ void recalcSigmaTotPar(Graph*g, float* sigmaTot, int* cliques) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int vertice = tid;
    if(tid < g->size){ // there is a chance that a whole block except one thread will be stuck but oh, well
        int clique = cliques[vertice];

        int edgesStart =  EDGES_IDX(g, vertice - 1);
        int edgesEnd =  EDGES_IDX(g, vertice);
        Edge * edgesPtr = g->edges + edgesStart;
        int numEdges = edgesEnd - edgesStart;
        float ki = getKiDevice(numEdges, edgesPtr);
        atomicAdd(sigmaTot + clique, ki);
    }
}

__global__ void calculateCliqueSizes(Graph*g, int* cliques, int * cliqueSizes) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < g->size){
        int vertice = tid;
        int clique = cliques[vertice];
        atomicAdd(cliqueSizes + clique, 1);
    }
}

__global__ void calcNeighbours(Graph *g, int *sizes){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int vertice = tid;
    if(tid < g->size){
        int edgesStart =  EDGES_IDX(g, vertice - 1);
        int edgesEnd =  EDGES_IDX(g, vertice);
        int numEdges = edgesEnd - edgesStart;
        sizes[tid] = numEdges;
    }
}



__global__ void calculateMoves(Graph *g, int* cliques, int*cliqueSizes,
        Move* moves, float m, float* sigmaTot,
        float minimum, int * nMoves){
    extern __shared__ float bestOutcomes[];
    int vertice = blockIdx.x;
    int edgesStart =  EDGES_IDX(g, vertice - 1);
    int edgesEnd =  EDGES_IDX(g, vertice);
    Edge * edgesPtr = g->edges + edgesStart;
    int numEdges = edgesEnd - edgesStart;
    int tid = threadIdx.x;
    bestOutcomes[tid + blockDim.x] = -1.f;
    bestOutcomes[tid] = 0;
    int cliqueFrom = cliques[vertice];
    if(tid < numEdges){
        Edge e = g->edges[tid];
        if(cliqueFrom != cliques[e.to]){
            int pretender = cliques[e.to];
            bestOutcomes[tid + blockDim.x] = __int2float_rn(pretender);
            bestOutcomes[tid] = dQDevice(g, vertice, cliques, pretender, sigmaTot, m, numEdges, edgesPtr);
        }
    }
    //reduce within a block
    for (int stride=1;stride<blockDim.x;stride*=2)
    {
        __syncthreads();
        if (tid%(2*stride)==0){
            if(bestOutcomes[tid] < bestOutcomes[tid+stride]){
                bestOutcomes[tid] = bestOutcomes[tid+stride];
                bestOutcomes[tid + blockDim.x] = bestOutcomes[tid + blockDim.x + stride];
            }
        }
    }
    if (tid==0){
        int toClique = __float2int_rn(bestOutcomes[blockDim.x]);
        float gain =  bestOutcomes[0];
        if(gain > minimum && moveValid(cliqueFrom, toClique, cliqueSizes)){
            int myMove = atomicAdd(nMoves, 1) - 1;
            Move m = {.vertice=vertice, .toClique = toClique, .gain=gain};
            moves[myMove] = m;
        }
    }
}




















int phaseOne(Graph *g, int *cliques, float minimum, float threshold){
    int changed = 1;
    int iters = 0;

    Graph * deviceGraph;
    copyGraphToDevice(g, &deviceGraph);

    float * deviceSigmaTot;
    HANDLE_ERROR(hipMalloc((void**) &deviceSigmaTot, sizeof(float) * g->size));
    thrust::device_ptr<float> deviceSigmaTot_ptr(deviceSigmaTot);
    thrust::fill(deviceSigmaTot_ptr, deviceSigmaTot_ptr + g->size, (float) 0);


    int * deviceCliques;
    copyArrayToDevice(cliques, g->size, &deviceCliques);

    recalcSigmaTotPar<<<(g->size + 255)/256, 256>>>(deviceGraph, deviceSigmaTot, deviceCliques);

    int * deviceCliqueSizes;
    HANDLE_ERROR(hipMalloc((void**) &deviceCliqueSizes, sizeof(int) * g->size));
    thrust::device_ptr<int> deviceCliqueSizes_ptr(deviceCliqueSizes);
    thrust::fill(deviceCliqueSizes_ptr, deviceCliqueSizes_ptr + g->size, (int) 0);

    int nMoves = g->size;

    int movesDone = 0;
    int * movesDoneDevice;
    HANDLE_ERROR(hipMalloc((void**) &movesDoneDevice, sizeof(int)));
    HANDLE_ERROR(hipMemcpy((void*) movesDoneDevice, (void*)&movesDone, sizeof(int), hipMemcpyHostToDevice));

    printf("alloc1 \n");

    int * deviceSizes;
    HANDLE_ERROR(hipMalloc((void**) &deviceSizes, sizeof(int) * g->size));
    thrust::device_ptr<int> deviceSizes_ptr(deviceSizes);
    thrust::fill(deviceSizes_ptr, deviceSizes_ptr + g->size, (float) 0);

    printf("alloc2 \n");

    calcNeighbours<<<(g->size + 255)/256, 256>>>(g, deviceSizes);

    printf("alloc3 \n");

    int maxNeighbours = thrust::reduce(deviceSizes_ptr, deviceSizes_ptr + g->size, (int) 0, thrust::maximum<int>());

    printf("reduce \n");

    float m = thrust::reduce(deviceSigmaTot_ptr, deviceSigmaTot_ptr + g->size, (float) 0, thrust::plus<float>());


    m = m/2;

    printf("calculated:\n");
    printf("m=%f\n, maxN=%d", m, maxNeighbours);

//    if(minimum < 1){
//        printf("exiting\n");
//        exit(10);
//
//    }

    float mod = modularity(g, cliques);



    while(changed != 0 ){

        Move empty = {.vertice=0,.toClique=0,.gain=0};
        thrust::device_vector<Move> deviceMoves(nMoves, empty);

        HANDLE_ERROR(hipMemcpy(deviceCliques, cliques, sizeof(int) * g->size, hipMemcpyHostToDevice));


        calculateCliqueSizes<<<(g->size + 255)/256, 256>>>(deviceGraph, deviceCliques, deviceCliqueSizes);

        if(DEBUG){
            printf("---------------------------- small iter %d ------------------------------------------\n", iters);
        }
        changed = 0;
        iters++;
        movesDone = 0;
        HANDLE_ERROR(hipMemcpy((void*) movesDoneDevice, (void*)&movesDone, sizeof(int), hipMemcpyHostToDevice));

        Move* deviceMovesPtr = thrust::raw_pointer_cast(&deviceMoves[0]);
        calculateMoves<<<g->size, maxNeighbours, maxNeighbours * 2 * sizeof(float)>>>(deviceGraph, deviceCliques, deviceCliqueSizes, deviceMovesPtr, m,deviceSigmaTot, minimum, movesDoneDevice);

        HANDLE_ERROR(hipMemcpy((void*)&movesDone, (void*) movesDoneDevice, sizeof(int), hipMemcpyDeviceToHost));

        if(DEBUG){
            printf("calculated %d moves\n", movesDone);
        }


        if(movesDone > 0){
            changed = 1;
        }
        //sort moves //TODO
//        thrust::stable_sort(deviceMoves.begin(),deviceMoves.end(), compareMovesThrust);


        // wydobyć, moves
//        HANDLE_ERROR(hipMemcpy((void*) cliques, (void*)deviceCliques, sizeof(int)*g->size, hipMemcpyDeviceToHost));

        Move * moves = (Move*) calloc(nMoves, sizeof(Move));

        //inaczej TODO

//        thrust::copy(deviceMoves.begin(), deviceMoves.end(), moves);
//        thrust::host_vector<int> H(deviceMoves.begin(), deviceMoves.end());
//        for(int cnt = 0; cnt < )
//        Move * deviceMovesPtr = thrust::raw_pointer_cast(deviceMoves);
        HANDLE_ERROR(hipMemcpy( moves, deviceMovesPtr, sizeof(Move)*g->size, hipMemcpyDeviceToHost));



        int movesToApply = calculateMovesToApply(1, movesDone, nMoves);

        int* newCliques = (int*) malloc(sizeof(int) * g->size);
        memcpy(newCliques, cliques, sizeof(int) * g->size);
        float newMod = previewModularity(g, newCliques, moves, movesDone, movesToApply, 1);

        if(DEBUG){
            printf("modularity gain if %d applied=%f\n",movesToApply, newMod - mod);
        }


        if(movesDone > 0){
            float bestdQ = moves[0].gain;
            int movesIter = 2;
            while((newMod - mod < threshold) && (movesToApply > 1 || bestdQ > threshold)){
                movesToApply = calculateMovesToApply(movesIter, movesDone, nMoves);
                memcpy(newCliques, cliques, sizeof(int) * g->size);
                newMod = previewModularity(g, newCliques, moves, movesDone, movesToApply, 0);
                movesIter++;
            }
            if (newMod - mod > threshold) {
                memcpy(cliques, newCliques, sizeof(int) * g->size);
                mod = newMod;
//                printf("%f, \n", modularity(g, cliques));
            }
            if(movesToApply == 1 && bestdQ < threshold){
                changed = 0;
            }
        } else {
            changed = 0;
        }

        free(moves);
        free(newCliques);
        if(changed != 0) {
            HANDLE_ERROR(hipMemcpy((void*)deviceCliques, (void*) cliques, sizeof(int)*g->size, hipMemcpyHostToDevice));
        }

        //distroy all ptrs
    }
    return iters;
}

/**
 * update all edges to go to and from superVertices aka cliques changed to vertices in phase2
 */
void changeEdges(Graph *g, const int *cliques){
    for(int i=0; i<g->numEdges; i++){
        Edge *e = g->edges + i;
        int vertice = e->from;
        int cliq = cliques[vertice];
        int superVertice = cliq;
        if(vertice != superVertice) {
            e->from = superVertice;
        }
        e->to = cliques[e->to];
    }
}

void changeCliqueToMin(Graph *g, int*cliques){
    int *mins = (int*) malloc(g->size* sizeof(int)); //minimalny wierzchołek w klice
    for(int i=0; i<g->size; i++){
        mins[i] = -1;
    }
    for(int vertice=0; vertice < g->size; vertice++){
        if(verticeHasEdges(g, vertice)){
            int cl = cliques[vertice];
            if(mins[cl] == -1 || mins[cl] > vertice){
                mins[cl] = vertice;
            }
        }
    }
    for(int i=0; i<g->size; i++){
        int cl = cliques[i];
        if(mins[cl] != -1){
            cliques[i] = mins[cl];
        }
    }
    free(mins);
}

void phaseTwo(Graph *g, int *cliques){
    changeCliqueToMin(g, cliques);
    changeEdges(g, cliques);
    sortEdges(g);

    float sum=0;
    int from=0;
    int to=0;
    //aggregate edges
    Edge *lastEdge = g->edges;
    for(int i=0; i<g->numEdges; i++){
        Edge *e = g->edges + i;
        if(e->from != from || e->to != to){
            if(e->from > from){
                from = e->from;
            }
            to = e->to;
            if(sum > 0){
                lastEdge->value = sum;
            }
            sum = 0;
        }
        lastEdge = e;
        sum += e->value;
        e->value = 0;
    }
    sortEdges(g);
}

/**
 * oldCliques represents vertice to clique mapping, this function updates it
 * when a super vertice changed place
 * @param size
 * @param oldCliques
 * @param newCliques
 */
void updateOldCliques(Graph *g, int* cliques){
    for(int i=0; i<g->size; i++){
        int index = cliques[i];
        while(index != cliques[index]){
            index = cliques[index];
        }
        if(cliques[i] != index){
            if(DEBUG){
            printf("CHANGED CLIQUE FOR %d FROM %d TO %d\n", i, cliques[i], index);
            }
            cliques[i] = index;
        }
    }
}

void printCliques(int size, int*cliques){
    printf("labs=[");
    for (int i = 0; i < size; ++i) {
//        printf("cliques[%d]=%d;\n", i, cliques[i]);
        printf("%d,", cliques[i]);
        if(i%500 == 0){
            printf("\n");
        }
    }
    printf("];\n");
}

void printUsage(char * name){
    printf("# Usage:\n");
    printf("# %s  [--verbose] <filename>\n", name);
    printf("#     --verbose   print out the links\n");
    printf("#     filename    name of file with MTX matrix\n");
}


int main(int argc, char **argv){
    char * fileName;
    int verbose = 0;
    if(argc < 2 || argc > 3){
        printf("wrong number of arguments!\n");
        printUsage(argv[0]);
        return 1;
    } else if(argc == 2){
        fileName = argv[1];
    } else {
        if(strcmp(argv[1], "--verbose") == 0){
            fileName = argv[2];
            verbose = 1;
        } else {
            printUsage(argv[0]);
            return 2;
        }
    }

    MData * dat = readData(fileName);

    Graph *g = initGraph(dat);
    destroyMData(dat);



    int* cliques = (int*) malloc(sizeof(int) * g->size);
    for(int i=0; i<g->size; i++){
        cliques[i]=i;
    }

    int bigLoopIteration = 0;
    float minimum = 0.1 / (2 + bigLoopIteration) - 0.02;

    float threshold = 0.00001f;

    // profiler at hangGlider_4 th=0.00001f

    float mod = modularity(g, cliques);
    printf("modularity:%f\n", mod);
    int iter = 10;
    while(iter > 1 || minimum > threshold/10.f){

//        printf("========= PHASE 1 ==================\n");
        minimum = 0.1 / (2 + bigLoopIteration) - 0.02;
        minimum = minimum < threshold/20.f ? threshold/20.f : minimum;
//        printf("min:%f\n", minimum);
        iter = phaseOne(g, cliques, minimum, threshold);

//        printCliques(g->size, cliques);

//        printf("========= PHASE 2 ==================\n");
        phaseTwo(g, cliques);
//        printEdges(g);
        updateOldCliques(g, cliques);
//        printf("modularity:%f\n", modularity(g, cliques));
//        printCliques(g->size, cliques);
        bigLoopIteration += 1;
    }
    printf("converged after %d iterations!\n", bigLoopIteration+1);
    if(verbose != 0){
        printCliques(g->size, cliques);
    }


    mod = modularity(g, cliques);
    printf("modularity:%f\n", mod);

    free(cliques);

    destroyGraph(g);



    return 0;
}