#include "hip/hip_runtime.h"
//
// Created by kajetan on 30.04.2020.
//
extern "C" {
    #include <assert.h>
    #include "louvain.h"
    #include "graph-utils.h"
}

#include "errors.h"
#include <thrust/fill.h>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

//__device__ float atomicAdd(float* address, float val)
//{
//    unsigned int* address_as_ull =
//            (unsigned int*)address;
//    unsigned int old = *address_as_ull, assumed;
//
//    do {
//        assumed = old;
//        old = atomicCAS(address_as_ull, assumed,
//                        __float_as_int(val +
//                                               __int_as_float(assumed)));
//
//        // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
//    } while (assumed != old);
//
//    return __int_as_float(old);
//}

__device__ int moveValid(int from, int to, int* cliqueSizes);

float getKi(Graph *g, int vertice){
    float sum=0;
    for(int i=EDGES_IDX(g,vertice-1); i<EDGES_IDX(g,vertice); i++){
        sum+= g->edges[i].value;
    }
    return sum;
}

float getKiin(Graph *g, int vertice, int* cliques, int in ){
    float sum=0;
    for(int i=EDGES_IDX(g,vertice-1); i<EDGES_IDX(g,vertice); i++){
        Edge e = g->edges[i];
        if(e.to != vertice && cliques[e.to] == in){
            sum+= e.value;
        }
    }
    return sum;
}

int bestClique(Graph *g, int vertice, int *cliques, float*sigmaTots, float m){
    float best = 0;
    int bestClique = -1;
    for(int i=EDGES_IDX(g,vertice-1); i<EDGES_IDX(g,vertice); i++){
        int to = g->edges[i].to;
        int in = cliques[to];
        if(in != bestClique && in != cliques[vertice]){
            float deltaQ =  0;//dQ(g, vertice, cliques, in, sigmaTots, m);
            if(deltaQ > best){
                best = deltaQ;
                bestClique = in;
            } else if (deltaQ == best){
                if(bestClique > in){
                    bestClique = in;
                }
            }
        }
    }
    return bestClique;
}

float selfLoop(Graph *g, int vertice){
    for(int k=EDGES_IDX(g,vertice-1); k<EDGES_IDX(g,vertice); k++){
        Edge e = g->edges[k];
        if(e.to == vertice){
            return e.value;
        }
    }
    return 0.0f;
}

int verticeHasEdges(Graph *g, int vertice){
    int has = EDGES_IDX(g, vertice-1) != EDGES_IDX(g, vertice);
    return has;
}

float modularity(Graph *g, int * cliques){
    float sum = 0;
    float m = 0;
    float * ac = (float*) calloc(g->size, sizeof(float));

    for(int i=0; i<g->size; i++){
        float ki = getKi(g, i);
        int clique = cliques[i];
        ac[clique] += ki;
        m += ki;
    }
    m = m/2.f;

    for(int i=0; i<g->size; i++){
        sum += ac[i] * ac[i];
    }

    sum = -sum/(2.f * m);

    for(int i=0; i < g->size; i++){
        float EiwCiBezi = getKiin(g, i, cliques, cliques[i]);
        sum += EiwCiBezi + selfLoop(g, i);
    }
    free(ac);
    return sum/(2.f*m);
}

int compareMoves( const void * a, const void * b){
    Move* ma = ((Move*)a);
    Move* mb = ((Move*)b);
    if(ma->gain > mb->gain) {
        return -1;
    } else if(mb->gain > ma->gain){
        return 1;
    } else {
        return 0;
    }
}

int compareMovesThrust( Move a, Move b){
    if(a.gain > b.gain) {
        return -1;
    } else if(b.gain > a.gain){
        return 1;
    } else {
        return 0;
    }
}

void applyBestMoves(int* cliques, Move* moves ,int nMoves, int nBest, int sort){
    if(nMoves == 0){
        return;
    }
    assert(nMoves >= nBest);
    if(nMoves != nBest && sort != 0){
        qsort(moves, nMoves, sizeof(Move), compareMoves);
    }
    for(int i=0; i < nBest; i++){
        Move m = moves[i];
        cliques[m.vertice] = m.toClique;
    }
}

float previewModularity(Graph * g, int*newCliques, Move* moves, int nMoves, int nBest, int sort){
    applyBestMoves(newCliques, moves, nMoves, nBest, sort);
    float newMod = modularity(g, newCliques);
    return newMod;
}


__device__ int moveValid(int from, int to, int* cliqueSizes){
    if(from == to){
        return 0;
    }
    if(from > to || cliqueSizes[from] > 1 || cliqueSizes[to] > 1){
        return 1;
    }
    return 0;
}



int calculateMovesToApply(int iters, int movesDone, int nMoves){
    int ret = movesDone;
    for(int i=0; i< iters; i++){
        ret = (ret + 1)/ 2;
    }
    return ret > 0 ? ret : 1;
}

__device__ float getKiDevice(int numEdges, Edge* edges){
    float sum = 0;
    for(int i=0; i<numEdges; i++){
        sum += edges[i].value;
    }
    return sum;
}

void copyGraphToDevice(Graph*g, Graph**deviceGraphPtr){

    Edge * edgesPtr ;
    int * vertPtr ;

    HANDLE_ERROR(hipMalloc((void**) &edgesPtr, sizeof(Edge) * g->numEdges));
    HANDLE_ERROR(hipMalloc((void**) &vertPtr, sizeof(int) * g->size));

//    printf("graph tables malloc succeded\n");


    HANDLE_ERROR(hipMemcpy((void*) edgesPtr, (void*)g->edges, sizeof(Edge) * g->numEdges, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy((void*) vertPtr, (void*)g->verticeLastEdgeExclusive, sizeof(int) * g->size, hipMemcpyHostToDevice));

//    printf("copying succeded\n");


    HANDLE_ERROR(hipMalloc((void**)deviceGraphPtr, sizeof(Graph)));

    Graph gr = {.size=g->size, .numEdges=g->numEdges, .edges=edgesPtr, .verticeLastEdgeExclusive=vertPtr};

    HANDLE_ERROR(hipMemcpy((void*)*deviceGraphPtr, (void*)&gr, sizeof(Graph), hipMemcpyHostToDevice));

//    printf("graph init succeded\n");
}

__device__ float getKiinDevice(Graph *g, int vertice, int* cliques, int in ){
    float sum=0;
    for(int i=EDGES_IDX(g,vertice-1); i<EDGES_IDX(g,vertice); i++){
        Edge e = g->edges[i];
        if(e.to != vertice && cliques[e.to] == in){
            sum+= e.value;
        }
    }
    return sum;
}



__device__ float dQDevice(Graph*g, int vertice, int *cliques, int in, float* sigmaTot, float m, int numEdges, Edge* edges){

    float ki = getKiDevice(numEdges, edges);
    float kiin = getKiinDevice(g, vertice, cliques, in);
    float EiwCiBezi = getKiinDevice(g, vertice, cliques, cliques[vertice]);
    float aciBezi= sigmaTot[cliques[vertice]] - ki;
    float acj = sigmaTot[in];
    float part1 = (kiin - EiwCiBezi)/m;
    float part2 = ki * (aciBezi - acj)/(2 * m * m);
    return  part1+part2;
}




void copyArrayToDevice(int * arr, int size, int** deviceArray){
    HANDLE_ERROR(hipMalloc((void**) deviceArray, sizeof(int) * size));
    HANDLE_ERROR(hipMemcpy((void*) *deviceArray, (void*)arr, sizeof(int) * size, hipMemcpyHostToDevice));
}

void copyFloatArrayToDevice(float * arr, int size, float** deviceArray){
    HANDLE_ERROR(hipMalloc((void**) deviceArray, sizeof(float) * size));
    HANDLE_ERROR(hipMemcpy((void*) *deviceArray, (void*)arr, sizeof(float) * size, hipMemcpyHostToDevice));
}

__global__ void recalcSigmaTotPar(Graph*g, float* sigmaTot, int* cliques) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int vertice = tid;
    if(tid < g->size){ // there is a chance that a whole block except one thread will be stuck but oh, well
        int clique = cliques[vertice];

        int edgesStart =  EDGES_IDX(g, vertice - 1);
        int edgesEnd =  EDGES_IDX(g, vertice);
        Edge * edgesPtr = g->edges + edgesStart;
        int numEdges = edgesEnd - edgesStart;
        float ki = getKiDevice(numEdges, edgesPtr);
        atomicAdd(sigmaTot + clique, ki);
    }
}

__global__ void calculateCliqueSizes(Graph*g, int* cliques, int * cliqueSizes) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < g->size){
        int vertice = tid;
        int clique = cliques[vertice];
        atomicAdd(cliqueSizes + clique, 1);
    }
}

__global__ void calcNeighbours(Graph *g, int *sizes){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int vertice = tid;
    if(vertice < g->size){
        int edgesStart =  EDGES_IDX(g, vertice - 1);
        int edgesEnd =  EDGES_IDX(g, vertice);
        int numEdges = edgesEnd - edgesStart;
        sizes[vertice] = numEdges;
    }
}



__global__ void calculateMoves(Graph *g, int* cliques, int*cliqueSizes,
        Move* moves, float m, float* sigmaTot,
        float minimum, int * nMoves){
    extern __shared__ float bestOutcomes[];
    int vertice = blockIdx.x;
    int edgesStart =  EDGES_IDX(g, vertice - 1);
    int edgesEnd =  EDGES_IDX(g, vertice);
    Edge * edgesPtr = g->edges + edgesStart;
    int numEdges = edgesEnd - edgesStart;
    int tid = threadIdx.x;
    bestOutcomes[tid + blockDim.x] = -1.f;
    bestOutcomes[tid] = 0;
    int cliqueFrom = cliques[vertice];
    if(tid < numEdges){
        Edge e = g->edges[edgesStart + tid];
        if(cliqueFrom != cliques[e.to]){
            int pretender = cliques[e.to];
            float deltaQ = dQDevice(g, vertice, cliques, pretender, sigmaTot, m, numEdges, edgesPtr);

            if(deltaQ > minimum && moveValid(cliqueFrom, pretender, cliqueSizes)){
                bestOutcomes[tid + blockDim.x] = __int2float_rn(pretender);
                bestOutcomes[tid] = deltaQ;
            }
        }
    }
    //reduce within a block
    for (int stride=1;stride<blockDim.x;stride*=2)
    {
        __syncthreads();
        if (tid%(2*stride)==0){
            if(tid+stride < blockDim.x && bestOutcomes[tid] < bestOutcomes[tid+stride]){
                bestOutcomes[tid] = bestOutcomes[tid+stride];
                bestOutcomes[tid + blockDim.x] = bestOutcomes[tid + blockDim.x + stride];
            }
        }
    }
    if (tid==0 && bestOutcomes[0] > 0){
        int toClique = __float2int_rn(bestOutcomes[blockDim.x]);
        float gain =  bestOutcomes[0];
        int myMove = atomicAdd(nMoves, 1) - 1;
        Move m = {.vertice=vertice, .toClique = toClique, .gain=gain};
        moves[myMove] = m;

    }
}



void destroyDeviceGraph(Graph * deviceGraph){
    Graph g;
    HANDLE_ERROR(hipMemcpy((void*) &g, (void*)deviceGraph, sizeof(Graph), hipMemcpyDeviceToHost));
    hipFree(g.edges);
    hipFree(g.verticeLastEdgeExclusive);
    hipFree(deviceGraph);
}
















int phaseOne(Graph *g, int *cliques, float minimum, float threshold){
    int changed = 1;
    int iters = 0;

    Graph * deviceGraph;
    copyGraphToDevice(g, &deviceGraph);

    float * deviceSigmaTot;
    HANDLE_ERROR(hipMalloc((void**) &deviceSigmaTot, sizeof(float) * g->size));
    thrust::device_ptr<float> deviceSigmaTot_ptr(deviceSigmaTot);
    thrust::fill(deviceSigmaTot_ptr, deviceSigmaTot_ptr + g->size, (float) 0);


    int * deviceCliques;
    copyArrayToDevice(cliques, g->size, &deviceCliques);

    recalcSigmaTotPar<<<(g->size + 255)/256, 256>>>(deviceGraph, deviceSigmaTot, deviceCliques);

    int * deviceCliqueSizes;
    HANDLE_ERROR(hipMalloc((void**) &deviceCliqueSizes, sizeof(int) * g->size));
    thrust::device_ptr<int> deviceCliqueSizes_ptr(deviceCliqueSizes);
    thrust::fill(deviceCliqueSizes_ptr, deviceCliqueSizes_ptr + g->size, (int) 0);

    int nMoves = g->size;

    int movesDone = 0;
    int * movesDoneDevice;
    HANDLE_ERROR(hipMalloc((void**) &movesDoneDevice, sizeof(int)));
    HANDLE_ERROR(hipMemcpy((void*) movesDoneDevice, (void*)&movesDone, sizeof(int), hipMemcpyHostToDevice));


    int * deviceSizes;
    HANDLE_ERROR(hipMalloc((void**) &deviceSizes, sizeof(int) * g->size));
    thrust::device_ptr<int> deviceSizes_ptr(deviceSizes);
    thrust::fill(deviceSizes_ptr, deviceSizes_ptr + g->size, (float) 0);


    calcNeighbours<<<(g->size + 255)/256, 256>>>(deviceGraph, deviceSizes);


    int maxNeighbours = thrust::reduce(deviceSizes_ptr, deviceSizes_ptr + g->size, (int) 0, thrust::maximum<int>());


    float m = thrust::reduce(deviceSigmaTot_ptr, deviceSigmaTot_ptr + g->size, (float) 0, thrust::plus<float>());


    m = m/2;
    if(DEBUG){
        printf("calculated:\n");
        printf("m=%f\n, maxN=%d", m, maxNeighbours);
    }


//    if(minimum < 1){
//        printf("exiting\n");
//        exit(10);
//
//    }

    float mod = modularity(g, cliques);

    if(DEBUG){
        printf("modularity: %f\n", mod);
    }


    while(changed != 0 ){

        Move empty = {.vertice=0,.toClique=0,.gain=0};


        Move * deviceMoves;
        HANDLE_ERROR(hipMalloc((void**) &deviceMoves, sizeof(Move) * nMoves));
        thrust::device_ptr<Move> deviceMoves_ptr(deviceMoves);
        thrust::fill(deviceMoves_ptr, deviceMoves_ptr + nMoves, empty);

        HANDLE_ERROR(hipMemcpy(deviceCliques, cliques, sizeof(int) * g->size, hipMemcpyHostToDevice));

        calculateCliqueSizes<<<(g->size + 255)/256, 256>>>(deviceGraph, deviceCliques, deviceCliqueSizes);

        thrust::fill(deviceSigmaTot_ptr, deviceSigmaTot_ptr + g->size, (float) 0);
        recalcSigmaTotPar<<<(g->size + 255)/256, 256>>>(deviceGraph, deviceSigmaTot, deviceCliques);

        if(DEBUG){
            printf("---------------------------- small iter %d ------------------------------------------\n", iters);
        }
        changed = 0;
        iters++;
        movesDone = 0;
        HANDLE_ERROR(hipMemcpy((void*) movesDoneDevice, (void*)&movesDone, sizeof(int), hipMemcpyHostToDevice));

        //todo change that to max threads 256 :)
        calculateMoves<<<g->size, maxNeighbours, maxNeighbours * 2 * sizeof(float)>>>(deviceGraph, deviceCliques, deviceCliqueSizes, deviceMoves, m,deviceSigmaTot, minimum, movesDoneDevice);

        HANDLE_ERROR(hipMemcpy((void*)&movesDone, (void*) movesDoneDevice, sizeof(int), hipMemcpyDeviceToHost));
        movesDone = movesDone > 0 ?  movesDone -1 : 0;

        if(DEBUG){
            printf("calculated %d moves\n", movesDone);
        }


        if(movesDone > 0){
            changed = 1;
        }
        //sort moves //TODO
//        thrust::stable_sort(deviceMoves.begin(),deviceMoves.end(), compareMovesThrust);

        if(DEBUG){
            printf("moves sorted\n");
        }

        Move * moves = (Move*) malloc(nMoves * sizeof(Move));

        //wydobyć moves
        HANDLE_ERROR(hipMemcpy( moves, deviceMoves, sizeof(Move)*nMoves, hipMemcpyDeviceToHost));

        if(DEBUG){
            printf("moves moved to host\n");
        }

        int movesToApply = calculateMovesToApply(1, movesDone, nMoves);

        int* newCliques = (int*) malloc(sizeof(int) * g->size);
        memcpy(newCliques, cliques, sizeof(int) * g->size);
        float newMod = previewModularity(g, newCliques, moves, movesDone, movesToApply, 1);

        if(DEBUG){
            printf("moves:\n");

            for(int i=0; i<movesDone; i++){
                Move m = moves[i];
                printf("move %d from %d to %d \tgain:%f\n", m.vertice, cliques[m.vertice], m.toClique, m.gain);
            }


            printf("modularity gain if %d applied=%f\n",movesToApply, newMod - mod);
        }


        if(movesDone > 0){
            Move bestMove = moves[0];
            float bestdQ = bestMove.gain;
            int movesIter = 2;
            while(bestdQ > 0 && (newMod - mod < threshold) && (movesToApply > 1 || bestdQ > threshold)){
                movesToApply = calculateMovesToApply(movesIter, movesDone, nMoves);
                memcpy(newCliques, cliques, sizeof(int) * g->size);
                newMod = previewModularity(g, newCliques, moves, movesDone, movesToApply, 0);

                if(DEBUG){
                    printf("modularity gain if %d applied=%f\n",movesToApply, newMod - mod);
                }

                movesIter++;
            }
            if (newMod - mod > threshold) {
                memcpy(cliques, newCliques, sizeof(int) * g->size);
                mod = newMod;
//                printf("modularity: %f\n", modularity(g, cliques));
            }
            if(movesToApply == 1 && bestdQ < threshold){
                changed = 0;
            }
        } else {
            changed = 0;
        }
        hipFree(deviceMoves);
        free(moves);
        free(newCliques);

    }

    hipFree(deviceSigmaTot);
    hipFree(deviceCliques);
    hipFree(deviceCliqueSizes);
    hipFree(movesDoneDevice);
    hipFree(deviceSizes);

    destroyDeviceGraph(deviceGraph);

    //distroy all ptrs and graph
    return iters;
}

/**
 * update all edges to go to and from superVertices aka cliques changed to vertices in phase2
 */
void changeEdges(Graph *g, const int *cliques){
    for(int i=0; i<g->numEdges; i++){
        Edge *e = g->edges + i;
        int vertice = e->from;
        int cliq = cliques[vertice];
        int superVertice = cliq;
        if(vertice != superVertice) {
            e->from = superVertice;
        }
        e->to = cliques[e->to];
    }
}

void changeCliqueToMin(Graph *g, int*cliques){
    int *mins = (int*) malloc(g->size* sizeof(int)); //minimalny wierzchołek w klice
    for(int i=0; i<g->size; i++){
        mins[i] = -1;
    }
    for(int vertice=0; vertice < g->size; vertice++){
        if(verticeHasEdges(g, vertice)){
            int cl = cliques[vertice];
            if(mins[cl] == -1 || mins[cl] > vertice){
                mins[cl] = vertice;
            }
        }
    }
    for(int i=0; i<g->size; i++){
        int cl = cliques[i];
        if(mins[cl] != -1){
            cliques[i] = mins[cl];
        }
    }
    free(mins);
}

void phaseTwo(Graph *g, int *cliques){
    changeCliqueToMin(g, cliques);
    changeEdges(g, cliques);
    sortEdges(g);

    float sum=0;
    int from=0;
    int to=0;
    //aggregate edges
    Edge *lastEdge = g->edges;
    for(int i=0; i<g->numEdges; i++){
        Edge *e = g->edges + i;
        if(e->from != from || e->to != to){
            if(e->from > from){
                from = e->from;
            }
            to = e->to;
            if(sum > 0){
                lastEdge->value = sum;
            }
            sum = 0;
        }
        lastEdge = e;
        sum += e->value;
        e->value = 0;
    }
    sortEdges(g);
}

/**
 * oldCliques represents vertice to clique mapping, this function updates it
 * when a super vertice changed place
 * @param size
 * @param oldCliques
 * @param newCliques
 */
void updateOldCliques(Graph *g, int* cliques){
    for(int i=0; i<g->size; i++){
        int index = cliques[i];
        while(index != cliques[index]){
            index = cliques[index];
        }
        if(cliques[i] != index){
            if(DEBUG){
            printf("CHANGED CLIQUE FOR %d FROM %d TO %d\n", i, cliques[i], index);
            }
            cliques[i] = index;
        }
    }
}

void printCliques(int size, int*cliques){
    printf("labs=[");
    for (int i = 0; i < size; ++i) {
//        printf("cliques[%d]=%d;\n", i, cliques[i]);
        printf("%d,", cliques[i]);
        if(i%500 == 0){
            printf("\n");
        }
    }
    printf("];\n");
}

void printUsage(char * name){
    printf("# Usage:\n");
    printf("# %s  -f mtx-matrix-file -g min-gain [-v]\n", name);
    printf("#     mtx-matrix-file   matrix in mtx format representing undirected weighted graph\n");
    printf("#     min-gain    minimal modularity gain to move a node between communities\n");
    printf("#     -v    verbose mode, printing communities\n");
}


int main(int argc, char **argv){
    char * fileName;
    int verbose = 0;
    float min_gain = 0;
    if(argc != 5 && argc != 6){
        printf("wrong number of arguments!\n");
        printUsage(argv[0]);
        return 1;
    } else if(strcmp(argv[1], "-f") && strcmp(argv[3], "-g")){
        if(argc == 6){
            if(strcmp(argv[5], "-v") != 0){
                printf("what is that gibberish?!\n");
                printUsage(argv[0]);
                return 1;
            }
            verbose = 1;
        }
        fileName = argv[2];
        min_gain = strtof(argv[4], NULL);
    } else {
        printf("what is that gibberish?!\n");
        printUsage(argv[0]);
        return 1;
    }

    MData * dat = readData(fileName);

    Graph *g = initGraph(dat);
    destroyMData(dat);



    int* cliques = (int*) malloc(sizeof(int) * g->size);
    for(int i=0; i<g->size; i++){
        cliques[i]=i;
    }

    int bigLoopIteration = 0;
    float minimum = 0.1 / (2 + bigLoopIteration) - 0.02;

    float threshold = min_gain;

    // profiler at hangGlider_4 th=0.00001f

    float mod = modularity(g, cliques);
//    printf("modularity:%f\n", mod);

    hipEvent_t start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start, 0));


    int iter = 10;
    while(iter > 1 || minimum > threshold/10.f){

        if(DEBUG){
            printf("========= PHASE 1 ==================\n");
        }
        minimum = 0.1 / (2 + bigLoopIteration) - 0.02;
        minimum = minimum < threshold/20.f ? threshold/20.f : minimum;
//        printf("min:%f\n", minimum);
        iter = phaseOne(g, cliques, minimum, threshold);
//        printCliques(g->size, cliques);
        if(DEBUG){
            printf("========= PHASE 2 ==================\n");
        }
        if(iter > 1){
            phaseTwo(g, cliques);
        }
//        printEdges(g);
        updateOldCliques(g, cliques);
//        printf("modularity:%f\n", modularity(g, cliques));
//        printCliques(g->size, cliques);
        bigLoopIteration += 1;
    }
    if(DEBUG){
        printf("converged after %d iterations!\n", bigLoopIteration+1);
    }
//    if(verbose != 0){
//        printCliques(g->size, cliques);
//    }


    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));

    float elapsedTime;
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

    printf("%f\n", modularity(g, cliques));

    printf("%f %f\n", elapsedTime, elapsedTime);

    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));

    free(cliques);

    destroyGraph(g);



    return 0;
}